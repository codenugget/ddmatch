// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define NX 256
#define BATCH 1

// Complex data type
typedef float2 Complex;

int main(int argc, char** argv) { 

  // Allocate host memory for the signal
  Complex *h_signal = reinterpret_cast<Complex *>(malloc(sizeof(Complex)*NX));

  // Initialize the memory for the signal
  for (unsigned int i = 0; i < NX; ++i) {
    h_signal[i].x = exp(-(double)i/NX);
    h_signal[i].y = 0;
  }

  hipfftHandle plan;
  hipfftComplex *data;
  
  hipMalloc((void**)&data, sizeof(hipfftComplex)*NX*BATCH);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to allocate\n");
    return -1;	
  }
  // Copy signal to device
  hipMemcpy(data, h_signal, sizeof(Complex)*NX, hipMemcpyHostToDevice);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to copy data to GPU\n");
    return -1;	
  }

  // perform Fourier transform
  if (hipfftPlan1d(&plan, NX, HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: Plan creation failed");
    return -1;
  }
  // Identical pointers to input and output arrays implies in-place transformation
  if (hipfftExecC2C(plan, data, data, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
    fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
    return -1;	
  }

  if (hipDeviceSynchronize() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to synchronize\n");
    return -1;	
  }	
  // Divide by number of elements in data set to get back original data


  hipMemcpy(h_signal, data, sizeof(Complex)*NX,
                             hipMemcpyDeviceToHost);
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr, "Cuda error: Failed to copy GPU data to host\n");
    return -1;	
  }

  // save to file
  // ...but for now, we print
  for (unsigned int i = 0; i < 10; ++i) {
    printf("h_signal[%d].x = %f\n", i, h_signal[i].x);
  }
  

  // cleanup memory
  free(h_signal);
  hipfftDestroy(plan);
  hipFree(data);
  //exit
  return 0;
}
